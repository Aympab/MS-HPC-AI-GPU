#include "hip/hip_runtime.h"
#include <math.h> // for M_PI = 3.1415....

#include "lbmFlowUtils.h"

#include "lbmFlowUtils_kernels.h"
#include "cuda_error.h"

// ======================================================
// ======================================================
void macroscopic(const LBMParams& params, 
                 const velocity_array_t v,
                 const real_t* fin_d,
                 real_t* rho_d,
                 real_t* ux_d,
                 real_t* uy_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  dim3 gridSize(nx/64,ny); 
  dim3 blockSize(64);

  macroscopic_kernel<<<gridSize, blockSize>>>(params,
                                              v,
                                              fin_d,
                                              rho_d,
                                              ux_d,
                                              uy_d);

  hipDeviceSynchronize();

} // macroscopic

// ======================================================
// ======================================================
void equilibrium(const LBMParams& params, 
                 const velocity_array_t v,
                 const weights_t t,
                 const real_t* rho_d,
                 const real_t* ux_d,
                 const real_t* uy_d,
                 real_t* feq_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  /*We want nx/64 * ny blocks, each with 64 threads
  each thread will do the loop on npop = 9
  */
  dim3 gridSize(nx/64,ny); 
  dim3 blockSize(64);

  // launch the kernel
  equilibrium_kernel<<<gridSize, blockSize>>>(params, v, t,
                                              rho_d,
                                              ux_d,
                                              uy_d,
                                              feq_d);

  hipDeviceSynchronize();
} // equilibrium

// ======================================================
// ======================================================
void init_obstacle_mask(const LBMParams& params, 
                        int* obstacle, 
                        int* obstacle_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  const real_t cx = params.cx;
  const real_t cy = params.cy;

  const real_t r = params.r;

  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {

      int index = i + nx * j;

      real_t x = 1.0*i;
      real_t y = 1.0*j;

      obstacle[index] = (x-cx)*(x-cx) + (y-cy)*(y-cy) < r*r ? 1 : 0;

    } // end for i
  } // end for j

  // copy host to device
  CUDA_API_CHECK( hipMemcpy( obstacle_d, obstacle, nx*ny * sizeof(int),
                            hipMemcpyHostToDevice ) );

  hipDeviceSynchronize();
} // init_obstacle_mask

// ======================================================
// ======================================================
__host__ __device__
real_t compute_vel(int dir, int i, int j, real_t uLB, real_t ly)
{

  // flow is along X axis
  // X component is non-zero
  // Y component is always zero

  return (1-dir) * uLB * (1 + 1e-4 * sin(j/ly*2*M_PI));

} // compute_vel

// ======================================================
// ======================================================
void initialize_macroscopic_variables(const LBMParams& params, 
                                      real_t* rho, real_t* rho_d,
                                      real_t* ux, real_t* ux_d,
                                      real_t* uy, real_t* uy_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {

      int index = i + nx * j;

      rho[index] = 1.0;
      ux[index]  = compute_vel(0, i, j, params.uLB, params.ly);
      uy[index]  = compute_vel(1, i, j, params.uLB, params.ly);

    } // end for i
  } // end for j

  // copy host to device
  CUDA_API_CHECK( hipMemcpy( rho_d, rho, nx*ny * sizeof(real_t),
                            hipMemcpyHostToDevice ) );

  CUDA_API_CHECK( hipMemcpy( ux_d, ux, nx*ny * sizeof(real_t),
                            hipMemcpyHostToDevice ) );

  CUDA_API_CHECK( hipMemcpy( uy_d, uy, nx*ny * sizeof(real_t),
                            hipMemcpyHostToDevice ) );

  hipDeviceSynchronize();
} // initialize_macroscopic_variables

// ======================================================
// ======================================================
void border_outflow(const LBMParams& params, real_t* fin_d)
{ 
  // const int nx = params.nx;
  const int ny = params.ny;

  /*Here we give 64 rows to 64 threads in 1 grid, they will all
  update one row
    fin[index1 + 6*nxny] = fin[index2 + 6*nxny];
    fin[index1 + 7*nxny] = fin[index2 + 7*nxny];
    fin[index1 + 8*nxny] = fin[index2 + 8*nxny];
  
  TODO : optimize this one by splitting both my row and cols
  */
  dim3 gridSize(ny/64); 
  dim3 blockSize(64);

  border_outflow_kernel<<<gridSize, blockSize>>>(params, fin_d);
  
  CUDA_KERNEL_CHECK("border_outflow_kernel");
  hipDeviceSynchronize();

} // border_outflow

// ======================================================
// ======================================================
void border_inflow(const LBMParams& params, const real_t* fin_d, 
                   real_t* rho_d, real_t* ux_d, real_t* uy_d)
{

  // TODO : call kernel
} // border_inflow

// ======================================================
// ======================================================
void update_fin_inflow(const LBMParams& params, const real_t* feq_d, 
                       real_t* fin_d)
{

  // TODO : call kernel

} // update_fin_inflow
  
// ======================================================
// ======================================================
void compute_collision(const LBMParams& params, 
                       const real_t* fin_d,
                       const real_t* feq_d,
                       real_t* fout_d)
{

  // const int nx = params.nx;
  // const int ny = params.ny;

  // TODO : call kernel

} // compute_collision

// ======================================================
// ======================================================
void update_obstacle(const LBMParams &params, 
                     const real_t* fin_d,
                     const int* obstacle_d, 
                     real_t* fout_d)
{

  // const int nx = params.nx;
  // const int ny = params.ny;

  // TODO : call kernel

} // update_obstacle

// ======================================================
// ======================================================
void streaming(const LBMParams& params,
               const velocity_array_t v,
               const real_t* fout_d,
               real_t* fin_d)
{

  // const int nx = params.nx;
  // const int ny = params.ny;

  // TODO : call kernel

} // streaming
